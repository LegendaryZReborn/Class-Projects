//Cavaughn Browne


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32


//function that will multiply the matrix with the vector by calling the kernel
//requires a matrix, vector and the resulting vector

void matrixVectorM(float A_h [][N], float B_h [][1], float C_h [][1]);

//kernel
__global__ void matrixVectorMKernel(float *A_d, float *B_d, float *C_d);

int main()
{
	float A_h[N][N];
	float B_h[N][1];	
	float C_h[N][1];
	float sum = 0;


	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
		{
			A_h[i][j] = 0.5;
		}
		
		B_h[i][0] = 1.5;
	}
	
	
	matrixVectorM(A_h, B_h, C_h);
	
	//sum all the elements of the result	

	for(int i = 0; i < N; i++)
	{
		sum += C_h[i][0];
	}
	
	printf("C[%d][%d] is %f\n", 0, 0, C_h[0][0]);
	printf("C[%d][%d] is %f\n", N - 1, 0, C_h[N-1][0]);
	printf("Sum of all the elements in the resulting vector is %f", sum);
	
	return 0;
}

void matrixVectorM(float A_h[][N], float B_h[][1], float C_h[][1])
{	
	int size = N * N * sizeof(float);
	int size2 = N * 1 * sizeof(float);
	float *A_d, *B_d, *C_d;

	//allocate memory for the data on the gpu
	hipMalloc((void**)&A_d, size);
	hipMalloc((void**)&B_d, size2);
	hipMalloc((void**)&C_d, size2);
	
	//copies existing data over to gpu from the host
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size2, hipMemcpyHostToDevice);

	//multiplies the matrix by the vector; uses 1 block of
	//32 threads.
	matrixVectorMKernel<<< 1, 32 >>>(A_d, B_d, C_d);
	
	//copy over the results back to the host
	hipMemcpy(C_h, C_d, size2, hipMemcpyDeviceToHost);
	
	//free gpu memory
	hipFree(A_d);
	hipFree(B_d);																											

	hipFree(C_d);

}


__global__ void matrixVectorMKernel(float *A_d, float *B_d, float *C_d)
{
	//each thread is the dot production of a row of the matrix
	// and the vector. each thread computes a row of the result
	int id, id2;
	C_d[threadIdx.x] = 0;
	for(int i = 0; i < N; i++)
	{
		id = threadIdx.x * N + i;
		C_d[threadIdx.x] += A_d[id] * B_d[i];

	}
	
}
