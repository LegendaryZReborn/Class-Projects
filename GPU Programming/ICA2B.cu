
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 64


void vectorAdd(int A_h[][N], int B_h[][N], int C_h[][N]);

__global__ void vectorAddKernel(int *A_d, int *B_d, int *C_d);

int main()
{
	int A_h[N][N];
	int B_h[N][N];	
	int C_h[N][N];


	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
		{
			A_h[i][j] = 2;
			B_h[i][j] = 1;
		}
	}
	
	vectorAdd(A_h, B_h, C_h);
	
	for(int r = 0; r < N; r++)
	{
		for(int c = 0; c < N; c++)
		{
			printf("C[%d][%d] is %d\n", r, c, C_h[r][c]);
		}
	}
	return 0;
}

void vectorAdd(int A_h[][N], int B_h[][N], int C_h[][N])
{	
	int size = N * N * sizeof(int);
	int *A_d, *B_d, *C_d;

	hipMalloc((void**)&A_d, size);
	hipMalloc((void**)&B_d, size);
	hipMalloc((void**)&C_d, size);
	
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	vectorAddKernel<<< 4, 1024 >>>(A_d, B_d, C_d);
	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(B_d);

	hipFree(C_d);

}


__global__ void vectorAddKernel(int *A_d, int *B_d, int *C_d)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	C_d[id] = A_d[id] + B_d[id];
}
