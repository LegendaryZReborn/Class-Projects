//GPU Programming - Project
//Name: Cavaughn Browne
//Parallel Programming Date: 12/5/2016

//Reads N sets of data from a file called data.dat and processes them using 
//the FFT-Cooley Tukey Algorithm. 

//compile with these lines with the data.dat file in the same directory
//module load cuda
//nvcc -arch=compute_35 -code=sm_35 CooleyTukeyImplementation.cu -o a.out



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


#define PI 3.141592653589793

struct ComplexNum 
{
	double real;
	double imag;
};

int main(int argc, char **argv)
{
	
	FILE *fp;

	fp = fopen("data.dat", "r");
	int N = atoi(argv[1]);
	struct ComplexNum Xk;
	struct ComplexNum x[N];
	struct ComplexNum XkResults[N];
	struct ComplexNum evenP;
	struct ComplexNum oddP;
	double c, s, realPart, imgPart;
	int m, k, j;
	float time;
	hipEvent_t start, stop;

	j = 0;
	
		while(j < N && !feof(fp))
		{
			fscanf(fp, "%lf", &x[j].real);
			fscanf(fp, "%lf", &x[j].imag);
			
			j++;
		}

	
	if(j < N)
	{
		for(; j < N; j++)
		{
			x[j].real = 0;
			x[j].imag = 0;
		}
	}

	printf("TOTAL PROCESSED SAMPLES: %d\n", N);

	hipEventCreate(&start);
	hipEventCreate(&stop) ;
	hipEventRecord(start, 0);
	
	clock_t st = clock(), diff;
	
	for (k = 0; k < N; k++)
	{
		double theta = (-2 * PI * k) / (N / 2);

		evenP.real = 0;
		evenP.imag = 0;
		oddP.real = 0;
		oddP.imag = 0;
		
		for ( m = 0; m < N / 2; m++)
		{
			c = cos(theta * m);
			s = sin(theta * m);
			
			//Even Indexed Part Computation
			realPart = (x[2 * m].real *c) - ((x[2 * m].imag * s));
			evenP.real += realPart;
			imgPart = (x[2 * m].real *s) + ((x[2 * m].imag * c));
			evenP.imag += imgPart;

			//Odd Indexed Part Computation
			realPart = (x[(2 * m) + 1].real *c) - ((x[(2 * m) + 1].imag * s));
			oddP.real += realPart;
			imgPart = (x[(2 * m) + 1].real *s) + ((x[(2 * m) + 1].imag * c));
			oddP.imag += imgPart;
		}

		//Add the real and the odd part sums and store the result.
		Xk.real = evenP.real + (cos(theta / 2) * oddP.real) - (sin(theta / 2) * oddP.imag);
		Xk.imag = evenP.imag + (cos(theta / 2) * oddP.imag) + (sin(theta / 2) * oddP.real);
		XkResults[k] = Xk;
	}
	
	//print first 8
	int p;
	for(p = 0; p < 8; p++)
	{
		printf("XR[%d] : %lf\nXI[%d] : %lf\n", p, XkResults[p].real, p, XkResults[p].imag);

	}
	
	diff = clock() - st;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	printf("Cuda Time:  %3.1f ms \n", time);
	printf("C Time: %3.1f ms \n", diff);

}