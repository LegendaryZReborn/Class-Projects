//Cavaughn Browne


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024


//function that will multiply the matrix with the vector by calling the kernel
//requires a matrix, vector and the resulting vector

float matrixVectorM(float A_h [][N], float B_h [][1], float C_h [][1]);

//kernel
__global__ void matrixVectorMKernel(float *A_d, float *B_d, float *C_d);

__global__ void addResult(float *C_d, float *sumR);
int main()
{
	float A_h[N][N];
	float B_h[N][1];	
	float C_h[N][1];
	float sum;


	for(int i = 0; i < N; i++)
	{
		for(int j = 0; j < N; j++)
		{
			A_h[i][j] = 0.5;
		}
		
		B_h[i][0] = 1.5;
	}
	
	//calculted the result and
	//sum all the elements of the result	
	sum = matrixVectorM(A_h, B_h, C_h);
	
	printf("C[%d][%d] is %f\n", 0, 0, C_h[0][0]);
	printf("C[%d][%d] is %f\n", N - 1, 0, C_h[N-1][0]);
	printf("Sum of all the elements in the resulting vector is %f", sum);
	
	return 0;
}

float matrixVectorM(float A_h[][N], float B_h[][1], float C_h[][1])
{	
	int size = N * N * sizeof(float);
	int size2 = N * 1 * sizeof(float);
	float *A_d, *B_d, *C_d;
	float sum [N];
	float *sumR;

	//allocate memory for the data on the gpu
	hipMalloc((void**)&A_d, size);
	hipMalloc((void**)&B_d, size2);
	hipMalloc((void**)&C_d, size2);
	hipMalloc((void**)&sumR, size2);

	
	//copies existing data over to gpu from the host
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size2, hipMemcpyHostToDevice);

	
	//multiplies the matrix by the vector; uses 4 block of
	//1024 threads.
	matrixVectorMKernel<<< 4, 256 >>>(A_d, B_d, C_d);
	
	//copy over the results back to the host
	hipMemcpy(C_h, C_d, size2, hipMemcpyDeviceToHost);
	hipMemcpy(sumR, C_d, size2, hipMemcpyDeviceToDevice);
	
	addResult<<< 1, 1024 >>>(C_d, sumR);
	hipMemcpy(sum, sumR, size2, hipMemcpyDeviceToHost);
	
	
	//free gpu memory
	hipFree(A_d);
	hipFree(B_d);																											

	hipFree(C_d);
	hipFree(sumR);
	
	return sum[0];

}


__global__ void matrixVectorMKernel(float *A_d, float *B_d, float *C_d)
{
	//each thread is the dot product of a row of the matrix
	// and the vector. each thread computes a row of the result
	
	int id = (blockDim.x * blockIdx.x) + threadIdx.x;
	int aId;
	C_d[id] = 0;
	for(int r = 0; r < N; r++)
	{
		aId = id * N + r;
		C_d[id] += A_d[aId] * B_d[r];
	}
	
}

__global__ void addResult(float *C_d, float *sumR)

{
	//Reduction: Interleaved addressing referenced from the nvidia website
	for
	(int s=1; s < blockDim.x; s *= 2) {
	if (threadIdx.x % (2*s) == 0) {
	sumR[threadIdx.x] += sumR[threadIdx.x + s];
	}
	__syncthreads();
	}
}
