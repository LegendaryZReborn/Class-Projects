#include "hip/hip_runtime.h"
//Assigntment #1
//Name: Cavaughn Browne 
//Parallel Programming Date: 10/9/2016

//Reads N sets of data from a file called data.dat and processes them using 
//the FFT-Cooley Tukey Algorithm. When building with gcc compiler, you must use -lm 
//parameter to let the compiler know program is using math.h on Maverick

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


#define PI 3.141592653589793

struct ComplexNum 
{
	double real;
	double imag;
};

void FFT(ComplexNum* signals, int N);

__global__ void FFT_Kernel(ComplexNum* signals_d, ComplexNum* evenOddR, int N, int k);

__global__ void FFT_KernelSuper(ComplexNum*signals_d, ComplexNum* XkResults, ComplexNum* evenOddR, int N);

__global__ void sumEvenOddS(ComplexNum* evenOddR, int N, int k);

int main(int argc, char **argv)
{
	
	FILE *fp;

	fp = fopen("data.dat", "r");
	int N = atoi(argv[1]);
	int j;
	struct ComplexNum x[N];
	
	j = 0;
	
		while(j < N && !feof(fp))
		{
			fscanf(fp, "%lf", &x[j].real);
			fscanf(fp, "%lf", &x[j].imag);
			//printf("XR[%d]: %f\n", j, x[j].real);
			//printf("XI[%d]: %f\n", j, x[j].imag);
			j++;
		}
		
		
	

	
	if(j < N)
	{
		for(; j < N; j++)
		{
			x[j].real = 0;
			x[j].imag = 0;
		}
	}
	printf("TOTAL PROCESSED SAMPLES: %d\n", N);
	FFT(x, N);
	
}

void FFT(ComplexNum* signals, int N)
{
	int size = N * 2* sizeof(double);
	int size2 = (N/2) * 2* sizeof(double)* (N/2) * 2* sizeof(double);
	int threads;
	int blocks;
	ComplexNum* signals_d;
	ComplexNum* XkResults_d;
	ComplexNum* evenOddR_d;
	ComplexNum XkResults_h[N];


	float time;
	hipEvent_t start, stop;

	
	if(N < 1024)
	{
		threads = N % 1024;
		blocks = N/threads;
		
	}
	else if(N % 1024 == 0)
	{
		threads = 1024;
		blocks = N/threads;
		
	}
	else
	{
		threads = 1024;
		blocks = (N/threads) + 1;
		

	}
	hipMalloc((void**)&signals_d, size);
	hipMemcpy(signals_d, signals, size, hipMemcpyHostToDevice);
	
	hipMalloc((void**)&XkResults_d, size);
	hipMalloc((void**)&evenOddR_d, size2);
	
	hipEventCreate(&start);
	hipEventCreate(&stop) ;
	hipEventRecord(start, 0) ;

	clock_t st = clock(), diff;
	
	//call Kernel
	FFT_KernelSuper<<<blocks, threads>>>(signals_d, XkResults_d, evenOddR_d, N);
	
	
	diff = clock() - st;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	hipMemcpy(XkResults_h, XkResults_d, size, hipMemcpyDeviceToHost);
	
	hipFree(signals_d);
	hipFree(XkResults_d);
	hipFree(evenOddR_d);

	//print out the first 8
	int p;
	for(p = 0; p < 8; p++)
	{
		printf("XR[%d] : %lf\nXI[%d] : %lf\n", p, XkResults_h[p].real, p, XkResults_h[p].imag);	
	}
	
	
	printf("Cuda Time:  %3.1f ms \n", time);
	printf("C Time: %3.1f ms \n", diff);
	
}

__global__ void FFT_Kernel(ComplexNum* signals_d, ComplexNum* evenOddR, int N, int k)
{
	struct ComplexNum evenOddS;
	struct ComplexNum evenP;
	struct ComplexNum oddP;
	double c, s, realPart, imgPart;
	int m;
	
	
	//thread will compute Xk....k = threadIdx.x
	//int k
	m = blockIdx.x * blockDim.x + threadIdx.x;
	double theta = (-2 * PI * k) / (N / 2);
	 if(m < (N / 2))
	 {
		evenP.real = 0;
		evenP.imag = 0;
		oddP.real = 0;
		oddP.imag = 0;
		
		
		//Even
		c = cos(theta * m);
		s = sin(theta * m);
		realPart = (signals_d[2 * m].real *c) - ((signals_d[2 * m].imag * s));
		evenP.real += realPart;
		imgPart = (signals_d[2 * m].real *s) + ((signals_d[2 * m].imag * c));
		evenP.imag += imgPart;

		//Odd
		realPart = (signals_d[(2 * m) + 1].real *c) - ((signals_d[(2 * m) + 1].imag * s));
		oddP.real += realPart;
		imgPart = (signals_d[(2 * m) + 1].real *s) + ((signals_d[(2 * m) + 1].imag * c));
		oddP.imag += imgPart;
		
		evenOddS.real = evenP.real + (cos(theta / 2) * oddP.real) - (sin(theta / 2) * oddP.imag);
		evenOddS.imag = evenP.imag + (cos(theta / 2) * oddP.imag) + (sin(theta / 2) * oddP.real);


		evenOddR[(k * (N/2)) + m] = evenOddS;

	}
		
		

}


__global__ void FFT_KernelSuper(ComplexNum*signals_d, ComplexNum* XkResults, ComplexNum* evenOddR, int N)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	
	//Calculates N/2 even and odd parts. Sums the even and odd part each thread
	FFT_Kernel<<< 1, 1024 >>>(signals_d, evenOddR, N, k);
	hipDeviceSynchronize();

	sumEvenOddS <<<1, 1024 >>>(evenOddR, N, k);
	
	XkResults[k] = evenOddR[k * (N/2)];	
}

__global__ void sumEvenOddS(ComplexNum* evenOddR, int N, int k)
{
	//Reduction: Interleaved addressing referenced from the nvidia website
	int s;
	int tid = (k*(N/2)) + (blockDim.x * blockIdx.x + threadIdx.x);
	for(s=1; s < N/2; s *= 2) {
		if (threadIdx.x % (2*s) == 0) 
		{
			evenOddR[tid].real += evenOddR[tid + s].real;
			evenOddR[tid].imag += evenOddR[tid + s].imag;

		}
	__syncthreads();
	}
	
}
